#include<hip/hip_runtime.h>
#include<iostream>

#define SIZE 10000000
#define streamSize 500
#define streamNumber SIZE/streamSize

__global__ void primeKernel(long long int first, long long int last){

    for (;first <= last; first++){
        bool flag = 0;
        for (long long int j = 2;!flag && j <= ceil(__dsqrt_rd(first)); j++){
            if (!fmodf(first,j)){
                flag = 1;
            } 
        }
    }
}

__host__ int main(){
    
    hipSetDevice(0);

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t stream[streamNumber];

    hipEventRecord(start, 0);
    for (int i = 0; i < streamNumber; i++){
        hipStreamCreate(&stream[i]); 
    }
    for (int i = 0; i < SIZE/streamSize ; i++){
        primeKernel <<< 1, 1, 0, stream[i%streamNumber]>>>(i*streamSize, ((i+1)*streamSize)-1);
    }
    for (int i = 0; i < streamNumber; i++ ){
        hipStreamSynchronize(stream[i]);
    }
    for(int i = 0; i < streamNumber; i++){
        hipStreamDestroy(stream[i]);
    }
    hipEventRecord(stop, 0);
    
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time taken %3.1f ms\n", elapsedTime);
    hipDeviceReset();

    return 0;
}
